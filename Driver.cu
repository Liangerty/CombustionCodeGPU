#include "hip/hip_runtime.h"
#include "Driver.h"
#include "Define.h"
#include "DParameter.h"
#include "Initialize.h"
#include "InviscidScheme.cuh"
#include "ViscousScheme.cuh"
#include "Thermo.cuh"
#include "fmt/core.h"
#include "TimeAdvanceFunc.cuh"
#include "TemporalScheme.cuh"

#if MULTISPECIES == 1
#else
#include "Constants.h"
#endif

cfd::Driver::Driver(Parameter &parameter, Mesh &mesh_
#if MULTISPECIES == 1
    , ChemData &chem_data
#endif
) : myid(parameter.get_int("myid")), mesh(mesh_), parameter(parameter), bound_cond() {
  // Allocate the memory for every block
  for (integer blk = 0; blk < mesh.n_block; ++blk) {
    field.emplace_back(parameter, mesh[blk]);
  }

  initialize_basic_variables(parameter, mesh, field
#if MULTISPECIES == 1
      , chem_data
#endif
  );

  // The following code is used for GPU memory allocation
#ifdef GPU
  DParameter d_param(parameter
#if MULTISPECIES == 1
      , chem_data
#endif
  );
  hipMalloc(&param, sizeof(DParameter));
  hipMemcpy(param, &d_param, sizeof(DParameter), hipMemcpyHostToDevice);
  for (integer blk = 0; blk < mesh.n_block; ++blk) {
    field[blk].setup_device_memory(parameter, mesh[blk]);
  }
  bound_cond.initialize_bc_on_GPU(mesh_, field
#if MULTISPECIES == 1
      , chem_data.spec
#endif
  );
  hipMalloc(&inviscid_scheme, sizeof(InviscidScheme *));
  hipMalloc(&viscous_scheme, sizeof(ViscousScheme *));
  hipMalloc(&temporal_scheme, sizeof(TemporalScheme *));

  setup_schemes<<<1, 1>>>(inviscid_scheme, viscous_scheme, temporal_scheme, param);
#endif
}

void cfd::Driver::initialize_computation() {
  dim3 tpb{8, 8, 4};
  if (mesh.dimension == 2) {
    tpb = {16, 16, 1};
  }
  const auto ng_1 = 2 * mesh[0].ngg - 1;

  // First, compute the conservative variables from basic variables
  for (auto i = 0; i < mesh.n_block; ++i) {
    integer mx{mesh[i].mx}, my{mesh[i].my}, mz{mesh[i].mz};
    dim3 bpg{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, (mz + ng_1) / tpb.z + 1};
    compute_cv_from_bv<<<bpg, tpb>>>(field[i].d_ptr, param);
  }

  // Second, apply boundary conditions to all boundaries, including face communication between faces
  bound_cond.apply_boundary_conditions(mesh, field, param);
  hipDeviceSynchronize();
  if (myid == 0) {
    fmt::print("Boundary conditions are applied successfully for initialization\n");
  }

  // Third, communicate values between processes
  // Currently not implemented, thus the current program can only be used on a single GPU

  if (myid == 0) {
    fmt::print("Finish data transfer.\n");
  }

  for (auto b = 0; b < mesh.n_block; ++b) {
    integer mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
    dim3 bpg{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, (mz + ng_1) / tpb.z + 1};
    update_physical_properties<<<bpg, tpb>>>(field[b].d_ptr, param);
  }
  hipDeviceSynchronize();
  if (myid == 0) {
    fmt::print("The flowfield is completely initialized on GPU.\n");
  }
}

void cfd::Driver::simulate() {
  const auto steady{parameter.get_bool("steady")};
  if (steady) {
    steady_simulation();
  } else {
    const auto temporal_scheme{parameter.get_int("temporal_scheme")};
    switch (temporal_scheme) {
      case 11:
      case 12:
      default:fmt::print("Not implemented");
    }
  }
}

void cfd::Driver::steady_simulation() {
  fmt::print("Steady flow simulation.\n");
  bool converged{false};
  integer step{0};
  integer total_step{parameter.get_int("total_step")};
  const integer n_block{mesh.n_block};
  const integer n_var{parameter.get_int("n_var")};

  dim3 tpb{8, 8, 4};
  if (mesh.dimension == 2) {
    tpb = {16, 16, 1};
  }

  //  const integer file_step{parameter.get_int("output_file")};
  while (!converged) {
    ++step;
    /*[[unlikely]]*/if (step > total_step) {
      break;
    }

    // Start a single iteration

    // First, store the value of last step
    for (auto b = 0; b < n_block; ++b) {
      const auto mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
      dim3 bpg{(mx - 1) / tpb.x + 1, (my - 1) / tpb.y + 1, (mz - 1) / tpb.z + 1};
      store_last_step<<<bpg, tpb>>>(field[b].d_ptr);
    }

    // Second, for each block, compute the residual dq
    for (auto b = 0; b < n_block; ++b) {
      compute_inviscid_flux(mesh[b], field[b].d_ptr, inviscid_scheme, param, n_var);
      compute_viscous_flux(mesh[b],field[b].d_ptr,viscous_scheme,param,n_var);
      compute_local_time_step(mesh[b],field[b].d_ptr,param,temporal_scheme);
      update_conservative_variables(mesh[b],field[b].d_ptr,param);
    }
    // Third, update conservative variables and apply boundary conditions.

    hipDeviceSynchronize();
    fmt::print("Step {}\n",step);
  }
}

__global__ void cfd::setup_schemes(cfd::InviscidScheme **inviscid_scheme, cfd::ViscousScheme **viscous_scheme,
                                   cfd::TemporalScheme **temporal_scheme, cfd::DParameter *param) {
  const integer inviscid_tag{param->inviscid_scheme};
  switch (inviscid_tag) {
    case 3:*inviscid_scheme = new AUSMP(param);
      printf("Inviscid scheme: AUSM+\n");
      break;
    default:*inviscid_scheme = new AUSMP(param);
      printf("No such scheme. Set to AUSM+ scheme\n");
  }

  const integer viscous_tag{param->viscous_scheme};
  switch (viscous_tag) {
    case 2:*viscous_scheme = new SecOrdViscScheme;
      printf("Viscous scheme: 2nd order central difference\n");
      break;
    default:*viscous_scheme = new ViscousScheme;
      printf("Inviscid computaion\n");
  }

  const integer temporal_tag{param->temporal_scheme};
  switch (temporal_tag) {
    case 0:
      *temporal_scheme=new SteadyTemporalScheme;
      printf("Temporal scheme: 1st order explicit Euler\n");
      break;
    case 1:
      *temporal_scheme=new LUSGS;
      printf("Temporal scheme: Implicit LUSGS\n");
      break;
    default:
      *temporal_scheme=new SteadyTemporalScheme;
      printf("Temporal scheme: 1st order explicit Euler\n");
  }
}
