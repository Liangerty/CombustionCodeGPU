#include "hip/hip_runtime.h"
#include "DataCommunication.cuh"
#include "Field.h"
#include "Constants.h"

//template<MixtureModel mix_model, TurbMethod turb_method>
//__global__ void cfd::inner_communication(DZone *zone, DZone *tar_zone, integer i_face) {
//  const auto &f = zone->innerface[i_face];
//  uint n[3];
//  n[0] = blockIdx.x * blockDim.x + threadIdx.x;
//  n[1] = blockDim.y * blockIdx.y + threadIdx.y;
//  n[2] = blockIdx.z * blockDim.z + threadIdx.z;
//  if (n[0] >= f.n_point[0] || n[1] >= f.n_point[1] || n[2] >= f.n_point[2]) return;
//
//  integer idx[3], idx_tar[3], d_idx[3];
//  for (integer i = 0; i < 3; ++i) {
//    d_idx[i] = f.loop_dir[i] * (integer) (n[i]);
//    idx[i] = f.range_start[i] + d_idx[i];
//  }
//  for (integer i = 0; i < 3; ++i) {
//    idx_tar[i] = f.target_start[i] + f.target_loop_dir[i] * d_idx[f.src_tar[i]];
//  }
//
//  // The face direction: which of i(0)/j(1)/k(2) is the coincided face.
//  const auto face_dir{f.direction > 0 ? f.range_start[f.face] : f.range_end[f.face]};
//
//  if (idx[f.face] == face_dir) {
//    // If this is the corresponding face, then average the values from both blocks
////    for (integer l = 0; l < 6; ++l) {
////      const real ave =
////          0.5 * (tar_zone->bv(idx_tar[0], idx_tar[1], idx_tar[2], l) + zone->bv(idx[0], idx[1], idx[2], l));
////      zone->bv(idx[0], idx[1], idx[2], l) = ave;
////      tar_zone->bv(idx_tar[0], idx_tar[1], idx_tar[2], l) = ave;
////    }
//    for (int l = 0; l < zone->n_var; ++l) {
//      const real ave =
//          0.5 * (tar_zone->cv(idx_tar[0], idx_tar[1], idx_tar[2], l) + zone->cv(idx[0], idx[1], idx[2], l));
//      zone->cv(idx[0], idx[1], idx[2], l) = ave;
//      tar_zone->cv(idx_tar[0], idx_tar[1], idx_tar[2], l) = ave;
//    }
//    update_bv_1_point(zone,param,idx[0], idx[1], idx[2]);
////    for (int l = 0; l < zone->n_scal; ++l) {
////      // Be Careful! The flamelet case is different from here, should be pay extra attention!
////      real ave = 0.5 * (tar_zone->sv(idx_tar[0], idx_tar[1], idx_tar[2], l) + zone->sv(idx[0], idx[1], idx[2], l));
////      zone->sv(idx[0], idx[1], idx[2], l) = ave;
////      tar_zone->sv(idx_tar[0], idx_tar[1], idx_tar[2], l) = ave;
////    }
//  } else {
//    // Else, get the inner value for this block's ghost grid
//    for (int l = 0; l < 6; ++l) {
//      zone->bv(idx[0], idx[1], idx[2], l) = tar_zone->bv(idx_tar[0], idx_tar[1], idx_tar[2], l);
//    }
//    for (int l = 0; l < zone->n_scal; ++l) {
//      // Be Careful! The flamelet case is different from here, should be pay extra attention!
//      zone->sv(idx[0], idx[1], idx[2], l) = tar_zone->sv(idx_tar[0], idx_tar[1], idx_tar[2], l);
//      zone->cv(idx[0], idx[1], idx[2], l + 5) = tar_zone->cv(idx_tar[0], idx_tar[1], idx_tar[2], l + 5);
//    }
//  }
//}

__global__ void cfd::setup_data_to_be_sent(cfd::DZone *zone, integer i_face, real *data) {
  const auto &f = zone->parface[i_face];
  integer n[3];
  n[0] = blockIdx.x * blockDim.x + threadIdx.x;
  n[1] = blockDim.y * blockIdx.y + threadIdx.y;
  n[2] = blockIdx.z * blockDim.z + threadIdx.z;
  if (n[0] >= f.n_point[0] || n[1] >= f.n_point[1] || n[2] >= f.n_point[2]) return;

  integer idx[3];
  for (int ijk: f.loop_order) {
    idx[ijk] = f.range_start[ijk] + n[ijk] * f.loop_dir[ijk];
  }

  const integer n_var{zone->n_var}, ngg{zone->ngg};
  integer bias = n_var * (ngg + 1) * (n[f.loop_order[1]] * f.n_point[f.loop_order[2]] + n[f.loop_order[2]]);

  const auto &cv = zone->cv;
  for (integer l = 0; l < n_var; ++l) {
    data[bias + l] = cv(idx[0], idx[1], idx[2], l);
  }

  for (integer ig = 1; ig <= ngg; ++ig) {
    idx[f.face] -= f.direction;
    bias += n_var;
    for (integer l = 0; l < n_var; ++l) {
      data[bias + l] = cv(idx[0], idx[1], idx[2], l);
    }
  }
}

