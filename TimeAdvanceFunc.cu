#include "hip/hip_runtime.h"
#include "TimeAdvanceFunc.cuh"
#include "Field.h"
#include "Mesh.h"
#include "InviscidScheme.cuh"
#include "ViscousScheme.cuh"
#include "TemporalScheme.cuh"

__global__ void cfd::store_last_step(cfd::DZone *zone) {
  const integer mx{zone->mx}, my{zone->my}, mz{zone->mz};
  integer i = blockDim.x * blockIdx.x + threadIdx.x;
  integer j = blockDim.y * blockIdx.y + threadIdx.y;
  integer k = blockDim.z * blockIdx.z + threadIdx.z;
  if (i >= mx || j >= my || k >= mz) return;

  zone->bv_last(i, j, k, 0) = zone->bv(i, j, k, 0);
  zone->bv_last(i, j, k, 1) = zone->vel(i, j, k);
  zone->bv_last(i, j, k, 2) = zone->bv(i, j, k, 4);
  zone->bv_last(i, j, k, 3) = zone->bv(i, j, k, 5);
}

void
cfd::compute_inviscid_flux(const Block &block, cfd::DZone *zone, InviscidScheme **inviscid_scheme, DParameter *param,
                           const integer n_var) {
  const integer extent[3]{block.mx, block.my, block.mz};
  const integer ngg{block.ngg};
  const integer dim{extent[2] == 1 ? 2 : 3};
  constexpr integer block_dim=64;
  const integer n_computation_per_block=block_dim+2*ngg-1;
  const auto shared_mem=(block_dim*n_var // fc
                  +n_computation_per_block*(n_var+3+1))* sizeof(real); // pv[n_var]+metric[3]+jacobian

  for (auto dir = 0; dir < dim; ++dir) {
    integer tpb[3]{1, 1, 1};
    tpb[dir] = block_dim;
    integer bpg[3]{extent[0], extent[1], extent[2]};
    bpg[dir] = (extent[dir] - 1) / (tpb[dir] - 1) + 1;

    dim3 TPB(tpb[0], tpb[1], tpb[2]);
    dim3 BPG(bpg[0], bpg[1], bpg[2]);
    inviscid_flux_1d<<<BPG, TPB, shared_mem>>>(zone, inviscid_scheme, dir, extent[dir], param);
  }
}

__global__ void
cfd::inviscid_flux_1d(cfd::DZone *zone, InviscidScheme **inviscid_scheme, integer direction, integer max_extent,
                      DParameter *param) {
  integer labels[3]{0, 0, 0};
  labels[direction] = 1;
  const integer tid = threadIdx.x * labels[0] + threadIdx.y * labels[1] + threadIdx.z * labels[2];
  const integer block_dim = blockDim.x * blockDim.y * blockDim.z;
  const auto ngg{zone->ngg};
  const integer n_point = block_dim + 2 * ngg - 1;

  integer idx[3];
  idx[0] = ((integer) blockDim.x - labels[0]) * blockIdx.x + threadIdx.x;
  idx[1] = ((integer) blockDim.y - labels[1]) * blockIdx.y + threadIdx.y;
  idx[2] = ((integer) blockDim.z - labels[2]) * blockIdx.z + threadIdx.z;
  idx[direction] -= 1;
  if (idx[direction] >= max_extent) return;

  // load variables to shared memory
  extern __shared__ real s[];
  const auto n_var{zone->n_var};
  real *pv = s;
  real *metric = &pv[n_point * n_var];
  real *jac = &metric[n_point * 3];
  real *fc = &jac[n_point];


  const auto n_spec{zone->n_spec};
  //
  const integer i_shared = tid - 1 + ngg;
  for (auto l = 0; l < 5; ++l) { // 0-rho,1-u,2-v,3-w,4-p
    pv[i_shared * n_var + l] = zone->bv(idx[0], idx[1], idx[2], l);
  }
  for (auto l = 0; l < n_spec; ++l) { // 5+l - Y_l
    pv[i_shared * n_var + 5 + l] = zone->yk(idx[0], idx[1], idx[2], l);
  }
  for (auto l = 1; l < 4; ++l) {
    metric[i_shared * 3 + l - 1] = zone->metric(idx[0], idx[1], idx[2])(direction + 1, l);
  }
  jac[i_shared] = zone->jac(idx[0], idx[1], idx[2]);

  // ghost cells
  if (tid == 0) {
    // Responsible for the left (ngg-1) points
    for (auto i = 1; i < ngg; ++i) {
      const auto ig_shared = ngg - 1 - i;
      const integer g_idx[3]{idx[0] - i * labels[0], idx[1] - i * labels[1], idx[2] - i * labels[2]};

      for (auto l = 0; l < 5; ++l) { // 0-rho,1-u,2-v,3-w,4-p
        pv[ig_shared * n_var + l] = zone->bv(g_idx[0], g_idx[1], g_idx[2], l);
      }
      for (auto l = 0; l < n_spec; ++l) { // 5+l - Y_l
        pv[ig_shared * n_var + 5 + l] = zone->yk(g_idx[0], g_idx[1], g_idx[2], l);
      }
      for (auto l = 1; l < 4; ++l) {
        metric[ig_shared * 3 + l - 1] = zone->metric(g_idx[0], g_idx[1], g_idx[2])(direction + 1, l);
      }
      jac[ig_shared] = zone->jac(g_idx[0], g_idx[1], g_idx[2]);
    }
  }
  if (tid == block_dim - 1 || idx[direction] == max_extent - 1) {
    // Responsible for the right ngg points
    for (auto i = 1; i <= ngg; ++i) {
      const auto ig_shared = tid + i + ngg - 1;
      const integer g_idx[3]{idx[0] + i * labels[0], idx[1] + i * labels[1], idx[2] + i * labels[2]};

      for (auto l = 0; l < 5; ++l) { // 0-rho,1-u,2-v,3-w,4-p
        pv[ig_shared * n_var + l] = zone->bv(g_idx[0], g_idx[1], g_idx[2], l);
      }
      for (auto l = 0; l < n_spec; ++l) { // 5+l - Y_l
        pv[ig_shared * n_var + 5 + l] = zone->yk(g_idx[0], g_idx[1], g_idx[2], l);
      }
      for (auto l = 1; l < 4; ++l) {
        metric[ig_shared * 3 + l - 1] = zone->metric(g_idx[0], g_idx[1], g_idx[2])(direction + 1, l);
      }
      jac[ig_shared] = zone->jac(g_idx[0], g_idx[1], g_idx[2]);
    }
  }
  __syncthreads();

  (*inviscid_scheme)->compute_inviscid_flux(zone, pv, tid, param, fc, metric, jac);
  __syncthreads();

  if (tid > 0) {
    for (integer l = 0; l < n_var; ++l) {
      zone->dq(idx[0], idx[1], idx[2], l) -= fc[tid * n_var + l] - fc[(tid - 1) * n_var + l];
    }
  }
}

void cfd::compute_viscous_flux(const cfd::Block &block, cfd::DZone *zone, cfd::ViscousScheme **viscous_scheme,
                               cfd::DParameter *param, integer n_var) {
  const integer extent[3]{block.mx, block.my, block.mz};
  const integer ngg{block.ngg};
  const integer dim{extent[2] == 1 ? 2 : 3};
  constexpr integer block_dim=64;

  dim3 tpb{block_dim,1,1};
  dim3 bpg((extent[0]-1)/(block_dim-1)+1,extent[1],extent[2]);
  auto shared_mem=block_dim*n_var* sizeof(real);
  viscous_flux_fv<<<bpg, tpb, shared_mem>>>(zone, viscous_scheme, extent[0], param);

  tpb={1,block_dim,1};
  bpg=(extent[0],(extent[1]-1)/(block_dim-1)+1,extent[2]);
  viscous_flux_gv<<<bpg, tpb, shared_mem>>>(zone, viscous_scheme, extent[1], param);

  if (dim==3){
    tpb={1,1,block_dim};
    bpg=(extent[0],extent[1],(extent[2]-1)/(block_dim-1)+1);
    viscous_flux_hv<<<bpg, tpb, shared_mem>>>(zone, viscous_scheme, extent[2], param);
  }
}

__global__ void
cfd::viscous_flux_fv(cfd::DZone *zone, cfd::ViscousScheme **viscous_scheme, integer max_extent,
                     cfd::DParameter *param) {
  integer idx[3];
  idx[0] = ((integer) blockDim.x - 1) * blockIdx.x + threadIdx.x-1;
  idx[1] = (integer) (blockDim.y * blockIdx.y + threadIdx.y);
  idx[2] = (integer) (blockDim.z * blockIdx.z + threadIdx.z);
  if (idx[0] >= max_extent) return;
  const auto tid=threadIdx.x;
  const auto n_var{zone->n_var};

  extern __shared__ real s[];
  real* fv=s;

  (*viscous_scheme)->compute_fv(idx,zone,&fv[tid*n_var],param);
  __syncthreads();

  if (tid>0){
    for (integer l=0;l<n_var;++l)
      zone->dq(idx[0],idx[1],idx[2],l)+=fv[tid*n_var+l]-fv[(tid-1)*n_var+l];
  }
}

__global__ void cfd::viscous_flux_gv(cfd::DZone *zone, cfd::ViscousScheme **viscous_scheme, integer max_extent,
                                     cfd::DParameter *param) {
  integer idx[3];
  idx[0] = (integer) (blockDim.x * blockIdx.x + threadIdx.x);
  idx[1] = ((integer) blockDim.y - 1) * blockIdx.y + threadIdx.y-1;
  idx[2] = (integer) (blockDim.z * blockIdx.z + threadIdx.z);
  if (idx[1] >= max_extent) return;
  const auto tid=threadIdx.y;
  const auto n_var{zone->n_var};

  extern __shared__ real s[];
  real* gv=s;

  (*viscous_scheme)->compute_gv(idx,zone,&gv[tid*n_var],param);
  __syncthreads();

  if (tid>0){
    for (integer l=0;l<n_var;++l)
      zone->dq(idx[0],idx[1],idx[2],l)+=gv[tid*n_var+l]-gv[(tid-1)*n_var+l];
  }
}

__global__ void cfd::viscous_flux_hv(cfd::DZone *zone, cfd::ViscousScheme **viscous_scheme, integer max_extent,
                                     cfd::DParameter *param) {
  integer idx[3];
  idx[0] = (integer) (blockDim.x * blockIdx.x + threadIdx.x);
  idx[1] = (integer) (blockDim.y * blockIdx.y + threadIdx.y);
  idx[2] = ((integer) blockDim.z - 1) * blockIdx.z + threadIdx.z-1;
  if (idx[2] >= max_extent) return;
  const auto tid=threadIdx.z;
  const auto n_var{zone->n_var};

  extern __shared__ real s[];
  real* hv=s;

  (*viscous_scheme)->compute_hv(idx,zone,&hv[tid*n_var],param);
  __syncthreads();

  if (tid>0){
    for (integer l=0;l<n_var;++l)
      zone->dq(idx[0],idx[1],idx[2],l)+=hv[tid*n_var+l]-hv[(tid-1)*n_var+l];
  }
}

void cfd::compute_local_time_step(const cfd::Block &block, cfd::DZone *zone, cfd::DParameter *param, TemporalScheme **temporal_scheme) {
  const integer extent[3]{block.mx, block.my, block.mz};
  const integer ngg{block.ngg};
  const integer dim{extent[2] == 1 ? 2 : 3};

  dim3 tpb{8, 8, 4};
  if (dim == 2)
    tpb = {16, 16, 1};
  dim3 bpg{(extent[0]-1)/tpb.x+1,(extent[1]-1)/tpb.y+1,(extent[2]-1)/tpb.z+1};
  local_time_step<<<bpg,tpb>>>(zone,param,temporal_scheme);
}

__global__ void cfd::local_time_step(cfd::DZone *zone, cfd::DParameter *param, TemporalScheme **temporal_scheme) {
  const integer extent[3]{zone->mx, zone->my, zone->mz};
  const auto i=(integer)(blockDim.x*blockIdx.x+threadIdx.x);
  const auto j=(integer)(blockDim.y*blockIdx.y+threadIdx.y);
  const auto k=(integer)(blockDim.z*blockIdx.z+threadIdx.z);
  if (i>=extent[0]||j>=extent[1]||k>=extent[2]) return;

  (*temporal_scheme)->compute_time_step(zone, i, j, k, param);
}

void cfd::update_conservative_variables(const Block &block, cfd::DZone *zone, DParameter *param) {
  const integer extent[3]{block.mx, block.my, block.mz};
  const integer ngg{block.ngg};
  const integer dim{extent[2] == 1 ? 2 : 3};

  dim3 tpb{8, 8, 4};
  if (dim == 2)
    tpb = {16, 16, 1};
  dim3 bpg{(extent[0]-1)/tpb.x+1,(extent[1]-1)/tpb.y+1,(extent[2]-1)/tpb.z+1};
  update_cv<<<bpg,tpb>>>(zone,param);
}

__global__ void cfd::update_cv(cfd::DZone *zone, cfd::DParameter *param) {
  const integer extent[3]{zone->mx, zone->my, zone->mz};
  const auto i=(integer)(blockDim.x*blockIdx.x+threadIdx.x);
  const auto j=(integer)(blockDim.y*blockIdx.y+threadIdx.y);
  const auto k=(integer)(blockDim.z*blockIdx.z+threadIdx.z);
  if (i>=extent[0]||j>=extent[1]||k>=extent[2]) return;

  real dt_div_jac=zone->dt_local(i,j,k)/zone->jac(i,j,k);
  for (integer l=0;l<zone->n_var;++l)
    zone->cv(i,j,k,l)+=zone->dq(i,j,k,l)*dt_div_jac;
  if (extent[2]==1)
    zone->cv(i,j,k,3)=0;
}
