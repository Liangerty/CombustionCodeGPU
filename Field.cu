#include "hip/hip_runtime.h"
#include "Field.h"
#include "Mesh.h"
#include "BoundCond.h"
#include "Thermo.cuh"
#include "DParameter.h"
#include "Transport.cuh"

cfd::HZone::HZone(Parameter &parameter, const Block &block) {
  const integer mx{block.mx}, my{block.my}, mz{block.mz}, ngg{block.ngg};
  const integer n_var{parameter.get_int("n_var")};

  cv.resize(mx, my, mz, n_var, ngg);
  bv.resize(mx, my, mz, 6, ngg);
  mach.resize(mx, my, mz, ngg);
#if MULTISPECIES == 1
  const integer n_spec{parameter.get_int("n_spec")};
  yk.resize(mx, my, mz, n_spec, ngg);
#endif
#ifdef _DEBUG
  dbv_squared.resize(mx, my, mz, 4, 0);
  tempo_var.resize(mx, my, mz, 0);
  dq.resize(mx, my, mz, n_var, 0);
#endif
}

void cfd::HZone::initialize_basic_variables(const cfd::Parameter &parameter, const cfd::Block &block,
                                            const std::vector<Inflow> &inflows, const std::vector<real> &xs,
                                            const std::vector<real> &xe, const std::vector<real> &ys,
                                            const std::vector<real> &ye, const std::vector<real> &zs,
                                            const std::vector<real> &ze) {
  const auto n = inflows.size();
  std::vector<real> rho(n, 0), u(n, 0), v(n, 0), w(n, 0), p(n, 0), T(n, 0);
#if MULTISPECIES == 1
  const auto n_spec = parameter.get_int("n_spec");
  gxl::MatrixDyn<double> mass_frac{static_cast<int>(n), n_spec};
#endif // MULTISPECIES==1
  for (size_t i = 0; i < inflows.size(); ++i) {
    std::tie(rho[i], u[i], v[i], w[i], p[i], T[i]) = inflows[i].var_info();
#if MULTISPECIES == 1
    auto y_spec = inflows[i].yk;
    for (int k = 0; k < n_spec; ++k) {
      mass_frac(static_cast<int>(i), k) = y_spec[k];
    }
#endif
  }

  const int ngg{block.ngg};
  for (int i = -ngg; i < block.mx + ngg; ++i) {
    for (int j = -ngg; j < block.my + ngg; ++j) {
      for (int k = -ngg; k < block.mz + ngg; ++k) {
        size_t i_init{0};
        if (inflows.size() > 1) {
          for (size_t l = 1; l < inflows.size(); ++l) {
            if (block.x(i, j, k) >= xs[l] && block.x(i, j, k) <= xe[l]
                && block.y(i, j, k) >= ys[l] && block.y(i, j, k) <= ye[l]
                && block.z(i, j, k) >= zs[l] && block.z(i, j, k) <= ze[l]) {
              i_init = l;
              break;
            }
          }
        }
        bv(i, j, k, 0) = rho[i_init];
        bv(i, j, k, 1) = u[i_init];
        bv(i, j, k, 2) = v[i_init];
        bv(i, j, k, 3) = w[i_init];
        bv(i, j, k, 4) = p[i_init];
        bv(i, j, k, 5) = T[i_init];
#if MULTISPECIES == 1
        for (int l = 0; l < n_spec; ++l) {
          yk(i, j, k, l) = mass_frac(static_cast<int>(i_init), l);
        }
#endif // MULTISPECIES==1
      }
    }
  }
}

cfd::Field::Field(Parameter &parameter, const Block &block)
    : h_zone(parameter, block) {}

void cfd::Field::initialize_basic_variables(const cfd::Parameter &parameter, const cfd::Block &block,
                                            const std::vector<Inflow> &inflows, const std::vector<real> &xs,
                                            const std::vector<real> &xe, const std::vector<real> &ys,
                                            const std::vector<real> &ye, const std::vector<real> &zs,
                                            const std::vector<real> &ze) {
  h_zone.initialize_basic_variables(parameter, block, inflows, xs, xe, ys, ye, zs, ze);
}

void cfd::Field::setup_device_memory(const Parameter &parameter, const Block &b) {
  h_ptr = new DZone;
  h_ptr->mx = b.mx, h_ptr->my = b.my, h_ptr->mz = b.mz, h_ptr->ngg = b.ngg;

  h_ptr->x.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  hipMemcpy(h_ptr->x.data(), b.x.data(), sizeof(real) * h_ptr->x.size(), hipMemcpyHostToDevice);
  h_ptr->y.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  hipMemcpy(h_ptr->y.data(), b.y.data(), sizeof(real) * h_ptr->y.size(), hipMemcpyHostToDevice);
  h_ptr->z.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  hipMemcpy(h_ptr->z.data(), b.z.data(), sizeof(real) * h_ptr->z.size(), hipMemcpyHostToDevice);

  auto n_bound{b.boundary.size()};
  auto n_inner{b.inner_face.size()};
  auto n_par{b.parallel_face.size()};
  auto mem_sz = sizeof(Boundary) * n_bound;
  hipMalloc(&h_ptr->boundary, mem_sz);
  hipMemcpy(h_ptr->boundary, b.boundary.data(), mem_sz, hipMemcpyHostToDevice);
  mem_sz = sizeof(InnerFace) * n_inner;
  hipMalloc(&h_ptr->innerface, mem_sz);
  hipMemcpy(h_ptr->innerface, b.inner_face.data(), mem_sz, hipMemcpyHostToDevice);
  mem_sz = sizeof(ParallelFace) * n_par;
  hipMalloc(&h_ptr->parface, mem_sz);
  hipMemcpy(h_ptr->parface, b.parallel_face.data(), mem_sz, hipMemcpyHostToDevice);

  h_ptr->jac.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  hipMemcpy(h_ptr->jac.data(), b.jacobian.data(), sizeof(real) * h_ptr->jac.size(), hipMemcpyHostToDevice);
  h_ptr->metric.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  hipMemcpy(h_ptr->metric.data(), b.metric.data(), sizeof(gxl::Matrix<real, 3, 3, 1>) * h_ptr->metric.size(),
             hipMemcpyHostToDevice);

  h_ptr->n_var = parameter.get_int("n_var");
  h_ptr->cv.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->n_var, h_ptr->ngg);
  h_ptr->bv.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, 6, h_ptr->ngg);
  hipMemcpy(h_ptr->bv.data(), h_zone.bv.data(), sizeof(real) * h_ptr->bv.sz * 6, hipMemcpyHostToDevice);
  h_ptr->bv_last.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, 4, 0);
  h_ptr->vel.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  h_ptr->acoustic_speed.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  h_ptr->mach.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  h_ptr->mul.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
  h_ptr->conductivity.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
#if MULTISPECIES == 1
  h_ptr->n_spec = parameter.get_int("n_spec");
  h_ptr->yk.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->n_spec, h_ptr->ngg);
  hipMemcpy(h_ptr->yk.data(), h_zone.yk.data(), sizeof(real) * h_ptr->yk.sz * h_ptr->n_spec, hipMemcpyHostToDevice);
  h_ptr->rho_D.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->n_spec, h_ptr->ngg);
  h_ptr->gamma.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->ngg);
#endif // MULTISPECIES==1
  h_ptr->dq.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, h_ptr->n_var, 0);
  if (parameter.get_int("temporal_scheme") == 1) {//LUSGS
    h_ptr->inv_spectr_rad.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, 0);
    h_ptr->visc_spectr_rad.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, 0);
  }
  if (parameter.get_bool("steady")) { // steady simulation
    h_ptr->dt_local.allocate_memory(h_ptr->mx, h_ptr->my, h_ptr->mz, 0);
  }

  hipMalloc(&d_ptr, sizeof(DZone));
  hipMemcpy(d_ptr, h_ptr, sizeof(DZone), hipMemcpyHostToDevice);
}

__global__ void cfd::compute_cv_from_bv(cfd::DZone *zone, cfd::DParameter *param) {
  const integer ngg{zone->ngg}, mx{zone->mx}, my{zone->my}, mz{zone->mz};
  integer i = (integer) (blockDim.x * blockIdx.x + threadIdx.x) - ngg;
  integer j = (integer) (blockDim.y * blockIdx.y + threadIdx.y) - ngg;
  integer k = (integer) (blockDim.z * blockIdx.z + threadIdx.z) - ngg;
  if (i >= mx + ngg || j >= my + ngg || k >= mz + ngg) return;

  const auto &bv = zone->bv;
  auto &cv = zone->cv;
  const real rho = bv(i, j, k, 0);
  const real u = bv(i, j, k, 1);
  const real v = bv(i, j, k, 2);
  const real w = bv(i, j, k, 3);

  cv(i, j, k, 0) = rho;
  cv(i, j, k, 1) = rho * u;
  cv(i, j, k, 2) = rho * v;
  cv(i, j, k, 3) = rho * w;
#if MULTISPECIES == 1
  const integer n_spec{zone->n_spec};
  const auto &yk = zone->yk;
  for (auto l = 0; l < n_spec; ++l) {
    cv(i, j, k, 5 + l) = rho * yk(i, j, k, l);
  }
#endif // MULTISPECIES==1
  compute_total_energy(i, j, k, zone, param);
}

__global__ void cfd::update_physical_properties(cfd::DZone *zone, cfd::DParameter *param) {
  const integer ngg{zone->ngg}, mx{zone->mx}, my{zone->my}, mz{zone->mz};
  integer i = (integer) (blockDim.x * blockIdx.x + threadIdx.x) - ngg;
  integer j = (integer) (blockDim.y * blockIdx.y + threadIdx.y) - ngg;
  integer k = (integer) (blockDim.z * blockIdx.z + threadIdx.z) - ngg;
  if (i >= mx + ngg || j >= my + ngg || k >= mz + ngg) return;

  const real temperature{zone->bv(i, j, k, 5)};
#if MULTISPECIES == 1
  const integer n_spec{zone->n_spec};
  auto &yk = zone->yk;
  real mw{0}, cp_tot{0}, cv{0};
  real *cp = new real[n_spec];
  compute_cp(temperature, cp, param);
  for (auto l = 0; l < n_spec; ++l) {
    mw += yk(i, j, k, l) / param->mw[l];
    cp_tot += yk(i, j, k, l) * cp[l];
    cv += yk(i, j, k, l) * (cp[l] - R_u / param->mw[l]);
  }
  mw = 1 / mw;
  zone->gamma(i, j, k) = cp_tot / cv;
  zone->acoustic_speed(i, j, k) = std::sqrt(zone->gamma(i, j, k) * R_u * temperature / mw);
  compute_transport_property(i, j, k, temperature, mw, cp, param, zone);
  delete[] cp;
#else
  constexpr real c_temp{gamma_air * R_u / mw_air};
  const real pr = param->Pr;
  zone->acoustic_speed(i,j,k) = std::sqrt(c_temp * temperature);
  zone->mul(i, j, k) = Sutherland(temperature);
  zone->conductivity(i, j, k) = zone->mul(i, j, k) * c_temp / (gamma_air - 1) / pr;
#endif
  zone->mach(i, j, k) = zone->vel(i, j, k) / zone->acoustic_speed(i, j, k);
}


__global__ void cfd::inner_communication(cfd::DZone *zone, cfd::DZone *tar_zone, const uint *n_point, integer i_face) {
  uint n[3];
  n[0] = blockIdx.x * blockDim.x + threadIdx.x;
  n[1] = blockDim.y + blockIdx.y + threadIdx.y;
  n[2] = blockIdx.z * blockDim.z + threadIdx.z;
  if (n[0] >= n_point[0] || n[1] >= n_point[1] || n[2] >= n_point[2]) return;

  integer idx[3], idx_tar[3];
  const auto &f = zone->innerface[i_face];
  for (integer i = 0; i < 3; ++i) {
    auto d_idx = f.loop_dir[i] * (integer) (n[i]);
    idx[i] = f.range_start[i] + d_idx;
    idx_tar[i] = f.target_start[i] + f.target_loop_dir[i] * d_idx;
  }

  // The face direction: which of i(0)/j(1)/k(2) is the coincided face.
  const auto face_dir{f.direction > 0 ? f.range_start[f.face] : f.range_end[f.face]};

  if (idx[f.face] == face_dir) {
    // If this is the corresponding face, then average the values from both blocks
    for (integer l = 0; l < 6; ++l) {
      const real ave =
          0.5 * (tar_zone->bv(idx_tar[0], idx_tar[1], idx_tar[2], l) + zone->bv(idx[0], idx[1], idx[2], l));
      zone->bv(idx[0], idx[1], idx[2], l) = ave;
      tar_zone->bv(idx_tar[0], idx_tar[1], idx_tar[2], l) = ave;
    }
    for (int l = 0; l < zone->n_var; ++l) {
      const real ave =
          0.5 * (tar_zone->cv(idx_tar[0], idx_tar[1], idx_tar[2], l) + zone->cv(idx[0], idx[1], idx[2], l));
      zone->cv(idx[0], idx[1], idx[2], l) = ave;
      tar_zone->cv(idx_tar[0], idx_tar[1], idx_tar[2], l) = ave;
    }
#if MULTISPECIES == 1
    for (int l = 0; l < zone->n_spec; ++l) {
      real ave = 0.5 * (tar_zone->yk(idx_tar[0], idx_tar[1], idx_tar[2], l) + zone->yk(idx[0], idx[1], idx[2], l));
      zone->yk(idx[0], idx[1], idx[2], l) = ave;
      tar_zone->yk(idx_tar[0], idx_tar[1], idx_tar[2], l) = ave;
    }
#endif
  } else {
    // Else, get the inner value for this block's ghost grid
    for (int l = 0; l < 5; ++l) {
      zone->bv(idx[0], idx[1], idx[2], l) = tar_zone->bv(idx_tar[0], idx_tar[1], idx_tar[2], l);
      zone->cv(idx[0], idx[1], idx[2], l) = tar_zone->cv(idx_tar[0], idx_tar[1], idx_tar[2], l);
    }
    zone->bv(idx[0], idx[1], idx[2], 5) = tar_zone->bv(idx_tar[0], idx_tar[1], idx_tar[2], 5);
#if MULTISPECIES == 1
    for (int l = 0; l < zone->n_spec; ++l) {
      zone->yk(idx[0], idx[1], idx[2], l) = tar_zone->yk(idx_tar[0], idx_tar[1], idx_tar[2], l);
      zone->cv(idx[0], idx[1], idx[2], l + 5) = tar_zone->cv(idx_tar[0], idx_tar[1], idx_tar[2], l + 5);
    }
#endif
  }
}

__global__ void cfd::eliminate_k_gradient(cfd::DZone *zone) {
  const integer ngg{zone->ngg}, mx{zone->mx}, my{zone->my};
  integer i = (integer) (blockDim.x * blockIdx.x + threadIdx.x) - ngg;
  integer j = (integer) (blockDim.y * blockIdx.y + threadIdx.y) - ngg;
  if (i >= mx + ngg || j >= my + ngg) return;

  auto &bv = zone->bv;
#if MULTISPECIES == 1
  auto &Y = zone->yk;
  const integer n_spec = zone->n_spec;
#endif

  for (integer k = 1; k <= ngg; ++k) {
    for (int l = 0; l < 6; ++l) {
      bv(i, j, k, l) = bv(i, j, 0, l);
      bv(i, j, -k, l) = bv(i, j, 0, l);
    }
#if MULTISPECIES == 1
    for (int l = 0; l < n_spec; ++l) {
      Y(i, j, k, l) = Y(i, j, 0, l);
      Y(i, j, -k, l) = Y(i, j, 0, l);
    }
#endif
  }
}
